#include "hip/hip_runtime.h"
//#include "gpupot.h"
// #include <iostream>
#include <cstdio>
// #include <cutil.h>
#ifdef WITH_CUDA5
#  include <hip/hip_runtime_api.h>
#  define CUDA_SAFE_CALL checkCudaErrors
#else
#  include <cutil.h>
#endif
#include "cuda_pointer.h"
#define NTHREAD 128

#define PROFILE
#ifdef PROFILE
#include <sys/time.h>
static double get_wtime(){
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return tv.tv_sec + 1.e-6 * tv.tv_usec;
}
#else
static double get_wtime(){
	return 0.0;
}
#endif

static float2 float2_split(double x){
	const int shift = 20;
	float2 ret;
	x *= (1<<shift);
	double xi = (int)x;
	double xf = x - xi;
	ret.x = xi * (1./(1<<shift));
	ret.y = xf * (1./(1<<shift));
	return ret;
}
__device__ float2 float2_accum(float2 acc, float x){
	float tmp = acc.x + x;
	acc.y -= (tmp - acc.x) - x;
	acc.x = tmp;
	return acc;
}

__device__ float2 float2_regularize(float2 acc){
	float tmp = acc.x + acc.y;
	acc.y = acc.y -(tmp - acc.x);
	acc.x = tmp;
	return acc;
}

__device__ float2 float2_add(float2 a, float2 b){
  float tmp = a.x + b.x;
  a.y -= (tmp - a.x) - b.x - b.y;
  a.x = tmp;
  // a.x = a.x + b.x;
  // a.y = a.y + b.y;
  return a;
}

struct Particle{
	float2 pos[3];
	float mass;
	float pad;

	Particle(double x[3], double m){
		pos[0] = float2_split(x[0]);
		pos[1] = float2_split(x[1]);
		pos[2] = float2_split(x[2]);
		mass = (float)m;
	}
	Particle(int){
		pos[0].x = pos[0].y = pos[1].x = pos[1].y = pos[2].x = pos[2].y = mass = pad = 0.f;
	}
	__device__ Particle() {}
};

__global__ void pot_kernel(int n, int nblock, Particle *ptcl, float2 *phi){
	__shared__ Particle jpbuf[NTHREAD];
	int i = NTHREAD * blockIdx.x + threadIdx.x;
	Particle ip = ptcl[i];
	float2 phii = make_float2(0.f, 0.f);
	for(int j=0; j<n; j+= NTHREAD){
		__syncthreads();
		jpbuf[threadIdx.x] = ptcl[j + threadIdx.x];
		__syncthreads();
#pragma unroll 4
		for(int jj=0; jj<NTHREAD; jj++){
			// if(j+jj == i) continue;
			Particle &jp = jpbuf[jj];
			float dx = (jp.pos[0].x - ip.pos[0].x) + (jp.pos[0].y - ip.pos[0].y);
			float dy = (jp.pos[1].x - ip.pos[1].x) + (jp.pos[1].y - ip.pos[1].y);
			float dz = (jp.pos[2].x - ip.pos[2].x) + (jp.pos[2].y - ip.pos[2].y);
			float r2 = dx*dx + dy*dy + dz*dz;
			// if(r2==0.f) continue;
			float pij = jp.mass * rsqrtf(r2);
			// phii = float2_accum(phii, pij);
			if(r2 > 0.f) phii = float2_accum(phii, pij);
		}
		phii = float2_regularize(phii);
	}
	phi[i] = phii;

    // for(int j = nblock/2 + nblock%2; j>1; j = j/2 + j%2) {
    //   int offset = j%2;
    //   if(blockIdx.x<j-offset) phi[i] = float2_add(phi[i],phi[i + j*NTHREAD]);
    // }
    
    // for(int j = NTHREAD/2; j>1; j/= 2) {
    //   if(threadIdx.x<j) phi[i] = float2_add(phi[i],phi[i + j]);
    // }
}

extern "C" void gpupot(int n, double **star, double *pot) {

    int numGPU=0;
    hipGetDeviceCount(&numGPU);
    assert(numGPU>0);
    hipSetDevice(0);

	double t0 = get_wtime();

	cudaPointer <float2> phi;
	cudaPointer <Particle> ptcl;

	int ng = NTHREAD * (n/NTHREAD + (n%NTHREAD ? 1 : 0));

	phi.allocate(ng);
	ptcl.allocate(ng);

	// std::cout << n << " " << ng << std::endl;
	for(int i=0; i<n; i++){
		// ptcl_h[i] = Particle(x[i], m[i]);
		ptcl[i] = Particle(&star[i][1], star[i][0]);
	}
	for(int i=n; i<ng; i++){
		// ptcl_h[i] = Particle(0);
		ptcl[i] = Particle(0);
	}

	// hipMemcpy(ptcl_d, ptcl_h, ng * sizeof(Particle), hipMemcpyHostToDevice);
	ptcl.htod(ng);
	
	dim3 grid(ng/NTHREAD, 1, 1);
	dim3 threads(NTHREAD, 1, 1);
	int sharedMemSize = NTHREAD * sizeof(Particle);
	// pot_kernel <<<grid, threads, sharedMemSize >>> (n, ptcl_d, phi_d);
    pot_kernel <<<grid, threads, sharedMemSize >>> (n, ng/NTHREAD, ptcl, phi);

    // phi.dtoh(1);
    // double pot = (double)phi[0].x + (double)phi[0].y;
	// hipMemcpy(phi_h, phi_d, n * sizeof(float2), hipMemcpyDeviceToHost);
    phi.dtoh(n);
    *pot = 0;
    for(int i=0; i<n; i++){
		// pot[i] = (double)phi_h[i].x + (double)phi_h[i].y;
      *pot -= star[i][0]*((double)phi[i].x + (double)phi[i].y);
    }
    *pot /=2.;
    
	phi.free();
	ptcl.free();

	double t1 = get_wtime();
#ifdef PROFILE
	fprintf(stderr, "gpupot: %f sec\n", t1 - t0);
#endif
}

